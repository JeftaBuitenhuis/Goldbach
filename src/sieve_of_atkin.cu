// CUDA runtime
#include <hip/hip_runtime.h>
//#include <immintrin.h> // SIMD AVX intel library
//#include <ia32intrin.h> // SIMD SVML intel library
#include <atomic> // atomic
#include <iostream> // io
#include <thread> // Multi-threading
#include <vector> // Vectors
#include <math.h> // Mathematics functions
#include <tuple> // tuple

__global__ void atkin_conditions(unsigned long max, unsigned long x, bool *p){
    unsigned long y = threadIdx.x + blockDim.x*blockIdx.x;
    //printf("%d\n", y);

    // Condition 1
    unsigned long n = (4 * x * x) + (y * y);
    if (n <= max && (n % 12 == 1 || n % 12 == 5)){
        p[n] ^= true;
    }

    // Condition 2
    n = (3 * x * x) + (y * y);
    if (n <= max && n % 12 == 7){
        p[n] ^= true;
    }

    // Condition 3
    n = (3 * x * x) - (y * y);
    if (x > y && n <= max && n % 12 == 11){
        p[n] ^= true;
    }
}

class Goldbach{
    public:
        /**
         * Constructor for the Goldbach class
         * 
         * @param max the maximum value calculated
         * @param threads the amount of CPU threads used for calculations
        */
        Goldbach(unsigned long max, int threads, int g_threads){
            THREADS = threads;
            MAX = max;
            CACHE_P = new bool[max + 1];
            CACHE_G = new unsigned long[max];
            CURRENT_NUM = MAX;
            G_THREADS = g_threads;
        }
        // FUNCTIONS
        void gen_primes();
        void write_cache_p();
        void write_cache_g();
        void initialize_threads();
    private:
        // VARIABLES
        unsigned long MAX;
        int THREADS = 1;
        int G_THREADS = 1024;
        //int SECTIONS;
        std::atomic_long CURRENT_NUM = 2;
        //int CURRENT_NUM = 2;
        bool *CACHE_P;
        

        // FUNCTIONS
        void solve_goldbach(unsigned long);
        void balance_load();

        // OUTPUT
        unsigned long *CACHE_G;
};

//----------------------------------------------------// Goldbach

void Goldbach::solve_goldbach(unsigned long num){
    unsigned long i = 2;
    bool stop = (CACHE_P[i] && CACHE_P[num-i]);
    i--;
    while (!stop){
        i+=2;
        //std::cout << i << "\n";
        stop = (CACHE_P[i] && CACHE_P[num-i]);
    }
    CACHE_G[num] = num-i;
    CACHE_G[num + 1] = i;
} 

void Goldbach::initialize_threads(){
    std::vector<std::thread> thread_vector;

    for (int thread = 0; thread < this->THREADS; thread++){
        thread_vector.emplace_back([&](){balance_load();});
    }

    for(auto& t: thread_vector){
        t.join();
    }
}

// void BalanceLoad()
void Goldbach::balance_load(){
    unsigned long n = (CURRENT_NUM -= 2);

    while (n >= 4){
        solve_goldbach(n);
        n = (CURRENT_NUM -= 2);
    }
}


//----------------------------------------------------// Primes

void Goldbach::gen_primes(){
    bool *d_x;//*h_x, *d_x;
    int nblocks = 1, nthreads=G_THREADS;

    //CACHE_P = (bool *) malloc(MAX*sizeof(bool) + sizeof(bool));
    hipMalloc((void **) &d_x, MAX*sizeof(bool) + sizeof(bool));

    CACHE_P[2] = true;
    CACHE_P[3] = true;

    for (unsigned long i = 0; i*i < MAX; i++){
        atkin_conditions<<<nblocks,nthreads>>>(MAX, i, d_x);
    }

    hipMemcpy(CACHE_P,d_x,MAX, hipMemcpyDeviceToHost);


    // Mark all multiples
    // of squares as non-prime
    for (unsigned long r = 5; r * r <= MAX; r++) {
        if (CACHE_P[r]) {
            for (unsigned long i = r * r; i <= MAX; i += r * r)
                CACHE_P[i] = false;
        }
    }
    hipFree(d_x);
}

//----------------------------------------------------------------// Debug

void Goldbach::write_cache_p(){
    unsigned long amount = 0;
    for (unsigned long i = 0; i < MAX; i++){
        if (CACHE_P[i]) {
            amount++;
        }
        std::cout << i << " = " << CACHE_P[i] << "\n";
    }
    std::cout << amount << "\n";
}

void Goldbach::write_cache_g(){
    unsigned long i_one_high = 0;
    unsigned long i_two_high = 0;
    for (unsigned long i = 4; i < MAX; i += 2){
        //std::cout << i << " = " << CACHE_G[i] << " + " << CACHE_G[i+1]  << "\n";
        if (i_one_high < CACHE_G[i]){
            i_one_high = CACHE_G[i];
        }
        if (i_two_high < CACHE_G[i+1]){
            i_two_high = CACHE_G[i+1];
        }
    }
    std::cout << "i_one_h: " << i_one_high << ", i_two_h: " << i_two_high << "\n";
}

//---------------------------------------------------------------// Main

int main(int argc, char* argv[]){
    unsigned long max = std::strtoul(argv[1], NULL, 0);
    int threads = std::stoi(argv[2]);
    int g_threads = std::stoi(argv[3]);
    bool debug = std::stoi(argv[4]);
    if (debug){
        //get_device_data();
    }
    Goldbach goldbach(max, threads, g_threads);
    goldbach.gen_primes();
    goldbach.initialize_threads();
    //goldbach.write_cache_p();
    //goldbach.write_cache_g();
}